#include "hip/hip_runtime.h"
// **************************************************************************
//                                   amoeba.cu
//                             -------------------
//                          Trung Dac Nguyen (Northwestern)
//
//  Device code for acceleration of the amoeba pair style
//
// __________________________________________________________________________
//    This file is part of the LAMMPS Accelerator Library (LAMMPS_AL)
// __________________________________________________________________________
//
//    begin                :
//    email                : trung.nguyen@northwestern.edu
// ***************************************************************************

#if defined(NV_KERNEL) || defined(USE_HIP)

#include "lal_aux_fun1.h"
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#include "inttypes.h"
#define tagint int64_t
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif
#ifndef _DOUBLE_DOUBLE
_texture( pos_tex,float4);
_texture( q_tex,float);
#else
_texture_2d( pos_tex,int4);
_texture( q_tex,int2);
#endif

#else
#define pos_tex x_
#define q_tex q_
#ifdef LAMMPS_SMALLBIG
#define tagint int
#endif
#ifdef LAMMPS_BIGBIG
#define tagint long
#endif
#ifdef LAMMPS_SMALLSMALL
#define tagint int
#endif

#endif // defined(NV_KERNEL) || defined(USE_HIP)


#if (SHUFFLE_AVAIL == 0)

#define local_allocate_store_ufld()                                         \
    __local acctyp red_acc[6][BLOCK_PAIR];

#define store_answers_amoeba_tq(tq, ii, inum,tid, t_per_atom, offset, i,    \
                                tep)                                        \
  if (t_per_atom>1) {                                                       \
    red_acc[0][tid]=tq.x;                                                   \
    red_acc[1][tid]=tq.y;                                                   \
    red_acc[2][tid]=tq.z;                                                   \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      simdsync();                                                           \
      if (offset < s) {                                                     \
        for (int r=0; r<3; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    tq.x=red_acc[0][tid];                                                   \
    tq.y=red_acc[1][tid];                                                   \
    tq.z=red_acc[2][tid];                                                   \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    tep[i]=tq;                                                              \
  }

#define store_answers_tep(ufld, dufld, ii, inum,tid, t_per_atom, offset,    \
                          i, tep)                                           \
  if (t_per_atom>1) {                                                       \
    red_acc[0][tid]=ufld[0];                                                \
    red_acc[1][tid]=ufld[1];                                                \
    red_acc[2][tid]=ufld[2];                                                \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      simdsync();                                                           \
      if (offset < s) {                                                     \
        for (int r=0; r<3; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    ufld[0]=red_acc[0][tid];                                                \
    ufld[1]=red_acc[1][tid];                                                \
    ufld[2]=red_acc[2][tid];                                                \
    red_acc[0][tid]=dufld[0];                                               \
    red_acc[1][tid]=dufld[1];                                               \
    red_acc[2][tid]=dufld[2];                                               \
    red_acc[3][tid]=dufld[3];                                               \
    red_acc[4][tid]=dufld[4];                                               \
    red_acc[5][tid]=dufld[5];                                               \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      simdsync();                                                           \
      if (offset < s) {                                                     \
        for (int r=0; r<6; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    dufld[0]=red_acc[0][tid];                                               \
    dufld[1]=red_acc[1][tid];                                               \
    dufld[2]=red_acc[2][tid];                                               \
    dufld[3]=red_acc[3][tid];                                               \
    dufld[4]=red_acc[4][tid];                                               \
    dufld[5]=red_acc[5][tid];                                               \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 t;                                                              \
    t.x = diz*ufld[1] - diy*ufld[2] + qixz*dufld[1] - qixy*dufld[3] +       \
      (numtyp)2.0*qiyz*(dufld[2]-dufld[5]) + (qizz-qiyy)*dufld[4];          \
    t.y = dix*ufld[2] - diz*ufld[0] - qiyz*dufld[1] + qixy*dufld[4] +       \
      (numtyp)2.0*qixz*(dufld[5]-dufld[0]) + (qixx-qizz)*dufld[3];          \
    t.z = diy*ufld[0] - dix*ufld[1] + qiyz*dufld[3] - qixz*dufld[4] +       \
      (numtyp)2.0*qixy*(dufld[0]-dufld[2]) + (qiyy-qixx)*dufld[1];          \
    tep[i]=t;                                                               \
  }

#define store_answers_fieldp(_fieldp, ii, inum,tid, t_per_atom, offset, i,  \
                              fieldp)                                       \
  if (t_per_atom>1) {                                                       \
    red_acc[0][tid]=_fieldp[0];                                             \
    red_acc[1][tid]=_fieldp[1];                                             \
    red_acc[2][tid]=_fieldp[2];                                             \
    red_acc[3][tid]=_fieldp[3];                                             \
    red_acc[4][tid]=_fieldp[4];                                             \
    red_acc[5][tid]=_fieldp[5];                                             \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      simdsync();                                                           \
      if (offset < s) {                                                     \
        for (int r=0; r<6; r++)                                             \
          red_acc[r][tid] += red_acc[r][tid+s];                             \
      }                                                                     \
    }                                                                       \
    _fieldp[0]=red_acc[0][tid];                                             \
    _fieldp[1]=red_acc[1][tid];                                             \
    _fieldp[2]=red_acc[2][tid];                                             \
    _fieldp[3]=red_acc[3][tid];                                             \
    _fieldp[4]=red_acc[4][tid];                                             \
    _fieldp[5]=red_acc[5][tid];                                             \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 f, fp;                                                          \
    f.x = _fieldp[0];                                                       \
    f.y = _fieldp[1];                                                       \
    f.z = _fieldp[2];                                                       \
    fieldp[ii] = f;                                                         \
    fp.x = _fieldp[3];                                                      \
    fp.y = _fieldp[4];                                                      \
    fp.z = _fieldp[5];                                                      \
    fieldp[ii+inum] = fp;                                                   \
  }

#define store_answers_acc(f,energy,e_coul, virial, ii, inum, tid, t_per_atom, \
                        offset, eflag, vflag, ans, engv, ev_stride)         \
  if (t_per_atom>1) {                                                       \
    simd_reduce_add3(t_per_atom, red_acc, offset, tid, f.x, f.y, f.z);      \
    if (EVFLAG && (vflag==2 || eflag==2)) {                                 \
      if (eflag) {                                                          \
        simdsync();                                                         \
        simd_reduce_add2(t_per_atom, red_acc, offset, tid, energy, e_coul); \
      }                                                                     \
      if (vflag) {                                                          \
        simdsync();                                                         \
        simd_reduce_arr(6, t_per_atom, red_acc, offset, tid, virial);       \
      }                                                                     \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }                                                                         \
  if (EVFLAG && (eflag || vflag)) {                                         \
    int ei=BLOCK_ID_X;                                                      \
    if (eflag!=2 && vflag!=2) {                                             \
      if (eflag) {                                                          \
        simdsync();                                                         \
        block_reduce_add2(simd_size(), red_acc, tid, energy, e_coul);       \
        if (vflag) __syncthreads();                                         \
        if (tid==0) {                                                       \
          engv[ei]+=energy*(acctyp)0.5;                                     \
          ei+=ev_stride;                                                    \
          engv[ei]+=e_coul*(acctyp)0.5;                                     \
          ei+=ev_stride;                                                    \
        }                                                                   \
      }                                                                     \
      if (vflag) {                                                          \
        simdsync();                                                         \
        block_reduce_arr(6, simd_size(), red_acc, tid, virial);             \
        if (tid==0) {                                                       \
          for (int r=0; r<6; r++) {                                         \
            engv[ei]+=virial[r]*(acctyp)0.5;                                \
            ei+=ev_stride;                                                  \
          }                                                                 \
        }                                                                   \
      }                                                                     \
    } else if (offset==0 && ii<inum) {                                      \
      int ei=ii;                                                            \
      if (EVFLAG && eflag) {                                                \
        engv[ei]+=energy*(acctyp)0.5;                                       \
        ei+=inum;                                                           \
        engv[ei]+=e_coul*(acctyp)0.5;                                       \
        ei+=inum;                                                           \
      }                                                                     \
      if (EVFLAG && vflag) {                                                \
        for (int i=0; i<6; i++) {                                           \
          engv[ei]+=virial[i]*(acctyp)0.5;                                  \
          ei+=inum;                                                         \
        }                                                                   \
      }                                                                     \
    }                                                                       \
  }

#else // SHUFFLE_AVAIL == 1

#define local_allocate_store_ufld()

#define store_answers_amoeba_tq(tq, ii, inum,tid, t_per_atom, offset, i,    \
                          tep)                                              \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      tq.x += shfl_down(tq.x, s, t_per_atom);                               \
      tq.y += shfl_down(tq.y, s, t_per_atom);                               \
      tq.z += shfl_down(tq.z, s, t_per_atom);                               \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    tep[i]=tq;                                                              \
  }

#define store_answers_tep(ufld, dufld, ii, inum,tid, t_per_atom, offset,    \
                          i, tep)                                           \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      ufld[0] += shfl_down(ufld[0], s, t_per_atom);                         \
      ufld[1] += shfl_down(ufld[1], s, t_per_atom);                         \
      ufld[2] += shfl_down(ufld[2], s, t_per_atom);                         \
      dufld[0] += shfl_down(dufld[0], s, t_per_atom);                       \
      dufld[1] += shfl_down(dufld[1], s, t_per_atom);                       \
      dufld[2] += shfl_down(dufld[2], s, t_per_atom);                       \
      dufld[3] += shfl_down(dufld[3], s, t_per_atom);                       \
      dufld[4] += shfl_down(dufld[4], s, t_per_atom);                       \
      dufld[5] += shfl_down(dufld[5], s, t_per_atom);                       \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 t;                                                              \
    t.x = diz*ufld[1] - diy*ufld[2] + qixz*dufld[1] - qixy*dufld[3] +       \
      (numtyp)2.0*qiyz*(dufld[2]-dufld[5]) + (qizz-qiyy)*dufld[4];          \
    t.y = dix*ufld[2] - diz*ufld[0] - qiyz*dufld[1] + qixy*dufld[4] +       \
      (numtyp)2.0*qixz*(dufld[5]-dufld[0]) + (qixx-qizz)*dufld[3];          \
    t.z = diy*ufld[0] - dix*ufld[1] + qiyz*dufld[3] - qixz*dufld[4] +       \
      (numtyp)2.0*qixy*(dufld[0]-dufld[2]) + (qiyy-qixx)*dufld[1];          \
    tep[i]=t;                                                               \
  }

#define store_answers_fieldp(_fieldp, ii, inum, tid, t_per_atom, offset, i, \
                             fieldp)                                        \
  if (t_per_atom>1) {                                                       \
    for (unsigned int s=t_per_atom/2; s>0; s>>=1) {                         \
      _fieldp[0] += shfl_down(_fieldp[0], s, t_per_atom);                   \
      _fieldp[1] += shfl_down(_fieldp[1], s, t_per_atom);                   \
      _fieldp[2] += shfl_down(_fieldp[2], s, t_per_atom);                   \
      _fieldp[3] += shfl_down(_fieldp[3], s, t_per_atom);                   \
      _fieldp[4] += shfl_down(_fieldp[4], s, t_per_atom);                   \
      _fieldp[5] += shfl_down(_fieldp[5], s, t_per_atom);                   \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 f, fp;                                                          \
    f.x = _fieldp[0];                                                       \
    f.y = _fieldp[1];                                                       \
    f.z = _fieldp[2];                                                       \
    fieldp[ii] = f;                                                         \
    fp.x = _fieldp[3];                                                      \
    fp.y = _fieldp[4];                                                      \
    fp.z = _fieldp[5];                                                      \
    fieldp[ii+inum] = fp;                                                   \
  }

#if (EVFLAG == 1)

#define store_answers_acc(f,energy,e_coul, virial, ii, inum, tid, t_per_atom, \
                        offset, eflag, vflag, ans, engv, ev_stride)         \
  if (t_per_atom>1) {                                                       \
    simd_reduce_add3(t_per_atom, f.x, f.y, f.z);                            \
    if (vflag==2 || eflag==2) {                                             \
      if (eflag)                                                            \
        simd_reduce_add2(t_per_atom,energy,e_coul);                         \
      if (vflag)                                                            \
        simd_reduce_arr(6, t_per_atom,virial);                              \
    }                                                                       \
  }                                                                         \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }                                                                         \
  if (eflag || vflag) {                                                     \
    if (eflag!=2 && vflag!=2) {                                             \
      const int vwidth = simd_size();                                       \
      const int voffset = tid & (simd_size() - 1);                          \
      const int bnum = tid/simd_size();                                     \
      int active_subgs = BLOCK_SIZE_X/simd_size();                          \
      for ( ; active_subgs > 1; active_subgs /= vwidth) {                   \
        if (active_subgs < BLOCK_SIZE_X/simd_size()) __syncthreads();       \
        if (bnum < active_subgs) {                                          \
          if (eflag) {                                                      \
            simd_reduce_add2(vwidth, energy, e_coul);                       \
            if (voffset==0) {                                               \
              red_acc[6][bnum] = energy;                                    \
              red_acc[7][bnum] = e_coul;                                    \
            }                                                               \
          }                                                                 \
          if (vflag) {                                                      \
            simd_reduce_arr(6, vwidth, virial);                             \
            if (voffset==0)                                                 \
              for (int r=0; r<6; r++) red_acc[r][bnum]=virial[r];           \
          }                                                                 \
        }                                                                   \
                                                                            \
        __syncthreads();                                                    \
        if (tid < active_subgs) {                                           \
          if (eflag) {                                                      \
            energy = red_acc[6][tid];                                       \
            e_coul = red_acc[7][tid];                                       \
          }                                                                 \
          if (vflag)                                                        \
            for (int r = 0; r < 6; r++) virial[r] = red_acc[r][tid];        \
        } else {                                                            \
          if (eflag) energy = e_coul = (acctyp)0;                           \
          if (vflag) for (int r = 0; r < 6; r++) virial[r] = (acctyp)0;     \
        }                                                                   \
      }                                                                     \
                                                                            \
      if (bnum == 0) {                                                      \
        int ei=BLOCK_ID_X;                                                  \
        if (eflag) {                                                        \
          simd_reduce_add2(vwidth, energy, e_coul);                         \
          if (tid==0) {                                                     \
            engv[ei]+=energy*(acctyp)0.5;                                   \
            ei+=ev_stride;                                                  \
            engv[ei]+=e_coul*(acctyp)0.5;                                   \
            ei+=ev_stride;                                                  \
          }                                                                 \
        }                                                                   \
        if (vflag) {                                                        \
          simd_reduce_arr(6, vwidth, virial);                               \
          if (tid==0) {                                                     \
            for (int r=0; r<6; r++) {                                       \
              engv[ei]+=virial[r]*(acctyp)0.5;                              \
              ei+=ev_stride;                                                \
            }                                                               \
          }                                                                 \
        }                                                                   \
      }                                                                     \
    } else if (offset==0 && ii<inum) {                                      \
      int ei=ii;                                                            \
      if (eflag) {                                                          \
        engv[ei]+=energy*(acctyp)0.5;                                       \
        ei+=inum;                                                           \
        engv[ei]+=e_coul*(acctyp)0.5;                                       \
        ei+=inum;                                                           \
      }                                                                     \
      if (vflag) {                                                          \
        for (int i=0; i<6; i++) {                                           \
          engv[ei]+=virial[i]*(acctyp)0.5;                                  \
          ei+=inum;                                                         \
        }                                                                   \
      }                                                                     \
    }                                                                       \
  }

// EVFLAG == 0
#else

#define store_answers_acc(f,energy,e_coul, virial, ii, inum, tid, t_per_atom, \
                        offset, eflag, vflag, ans, engv, ev_stride)         \
  if (t_per_atom>1)                                                         \
    simd_reduce_add3(t_per_atom, f.x, f.y, f.z);                            \
  if (offset==0 && ii<inum) {                                               \
    acctyp4 old=ans[ii];                                                    \
    old.x+=f.x;                                                             \
    old.y+=f.y;                                                             \
    old.z+=f.z;                                                             \
    ans[ii]=old;                                                            \
  }

#endif // EVFLAG
#endif // SHUFFLE_AVAIL

#define MIN(A,B) ((A) < (B) ? (A) : (B))
#define MY_PIS (acctyp)1.77245385090551602729

/* ----------------------------------------------------------------------
   multipole_real = real-space portion of multipole
   adapted from Tinker emreal1d() routine
------------------------------------------------------------------------- */

__kernel void k_amoeba_multipole(const __global numtyp4 *restrict x_,
                                 const __global numtyp4 *restrict extra,
                                 const __global numtyp4 *restrict coeff,
                                 const __global numtyp4 *restrict sp_amoeba,
                                 const __global int *dev_nbor,
                                 const __global int *dev_packed,
                                 const __global int *dev_short_nbor,
                                 __global acctyp4 *restrict ans,
                                 __global acctyp *restrict engv,
                                 __global acctyp4 *restrict tep,
                                 const int eflag, const int vflag, const int inum,
                                 const int nall, const int nbor_pitch,
                                 const int t_per_atom, const numtyp aewald,
                                 const numtyp felec, const numtyp off2,
                                 const numtyp polar_dscale, const numtyp polar_uscale)
{
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_charge();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, e_coul, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int l=0; l<6; l++) virial[l]=(acctyp)0;
  }

  acctyp4 tq;
  tq.x=(acctyp)0; tq.y=(acctyp)0; tq.z=(acctyp)0;
  
  const __global numtyp4* polar1 = &extra[0];
  const __global numtyp4* polar2 = &extra[nall];
  const __global numtyp4* polar3 = &extra[2*nall];

  if (ii<inum) {
    int numj, nbor, nbor_end;
    const __global int* nbor_mem=dev_packed;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor];
      nbor += n_stride;
      nbor_end = nbor+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    const numtyp4 pol1i = polar1[i];
    numtyp ci  = pol1i.x;    // rpole[i][0];
    numtyp dix = pol1i.y;    // rpole[i][1];
    numtyp diy = pol1i.z;    // rpole[i][2];
    numtyp diz = pol1i.w;    // rpole[i][3];
    const numtyp4 pol2i = polar2[i];
    numtyp qixx = pol2i.x;   // rpole[i][4];
    numtyp qixy = pol2i.y;   // rpole[i][5];
    numtyp qixz = pol2i.z;   // rpole[i][6];
    numtyp qiyy = pol2i.w;   // rpole[i][8];
    const numtyp4 pol3i = polar3[i];
    numtyp qiyz = pol3i.x;   // rpole[i][9];
    numtyp qizz = pol3i.y;   // rpole[i][12];

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int jextra=nbor_mem[nbor];
      int j = jextra & NEIGHMASK15;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp xr = jx.x - ix.x;
      numtyp yr = jx.y - ix.y;
      numtyp zr = jx.z - ix.z;
      numtyp r2 = xr*xr + yr*yr + zr*zr;

      numtyp r = ucl_sqrt(r2);
      const numtyp4 pol1j = polar1[j];
      numtyp ck  = pol1j.x;  // rpole[j][0];
      numtyp dkx = pol1j.y;  // rpole[j][1];
      numtyp dky = pol1j.z;  // rpole[j][2];
      numtyp dkz = pol1j.w;  // rpole[j][3];
      const numtyp4 pol2j = polar2[j];
      numtyp qkxx = pol2j.x; // rpole[j][4];
      numtyp qkxy = pol2j.y; // rpole[j][5];
      numtyp qkxz = pol2j.z; // rpole[j][6];
      numtyp qkyy = pol2j.w; // rpole[j][8];
      const numtyp4 pol3j = polar3[j];
      numtyp qkyz = pol3j.x; // rpole[j][9];
      numtyp qkzz = pol3j.y; // rpole[j][12];
      //int jtype = pol3j.z; // amtype[j];
      //int jgroup =  pol3j.w; // amgroup[j];

      const numtyp4 sp_pol = sp_amoeba[sbmask15(jextra)];
      numtyp factor_mpole = sp_pol.w; // sp_mpole[sbmask15(jextra)];

      // intermediates involving moments and separation distance

      numtyp dir = dix*xr + diy*yr + diz*zr;
      numtyp qix = qixx*xr + qixy*yr + qixz*zr;
      numtyp qiy = qixy*xr + qiyy*yr + qiyz*zr;
      numtyp qiz = qixz*xr + qiyz*yr + qizz*zr;
      numtyp qir = qix*xr + qiy*yr + qiz*zr;
      numtyp dkr = dkx*xr + dky*yr + dkz*zr;
      numtyp qkx = qkxx*xr + qkxy*yr + qkxz*zr;
      numtyp qky = qkxy*xr + qkyy*yr + qkyz*zr;
      numtyp qkz = qkxz*xr + qkyz*yr + qkzz*zr;
      numtyp qkr = qkx*xr + qky*yr + qkz*zr;

      numtyp dik = dix*dkx + diy*dky + diz*dkz;
      numtyp qik = qix*qkx + qiy*qky + qiz*qkz;
      numtyp diqk = dix*qkx + diy*qky + diz*qkz;
      numtyp dkqi = dkx*qix + dky*qiy + dkz*qiz;
      numtyp qiqk = (numtyp)2.0*(qixy*qkxy+qixz*qkxz+qiyz*qkyz) +
        qixx*qkxx + qiyy*qkyy + qizz*qkzz;

      // additional intermediates involving moments and distance

      numtyp dirx = diy*zr - diz*yr;
      numtyp diry = diz*xr - dix*zr;
      numtyp dirz = dix*yr - diy*xr;
      numtyp dikx = diy*dkz - diz*dky;
      numtyp diky = diz*dkx - dix*dkz;
      numtyp dikz = dix*dky - diy*dkx;
      numtyp qirx = qiz*yr - qiy*zr;
      numtyp qiry = qix*zr - qiz*xr;
      numtyp qirz = qiy*xr - qix*yr;
      numtyp qikx = qky*qiz - qkz*qiy;
      numtyp qiky = qkz*qix - qkx*qiz;
      numtyp qikz = qkx*qiy - qky*qix;
      numtyp qixk = qixx*qkx + qixy*qky + qixz*qkz;
      numtyp qiyk = qixy*qkx + qiyy*qky + qiyz*qkz;
      numtyp qizk = qixz*qkx + qiyz*qky + qizz*qkz;
      numtyp qkxi = qkxx*qix + qkxy*qiy + qkxz*qiz;
      numtyp qkyi = qkxy*qix + qkyy*qiy + qkyz*qiz;
      numtyp qkzi = qkxz*qix + qkyz*qiy + qkzz*qiz;
      numtyp qikrx = qizk*yr - qiyk*zr;
      numtyp qikry = qixk*zr - qizk*xr;
      numtyp qikrz = qiyk*xr - qixk*yr;
      numtyp diqkx = dix*qkxx + diy*qkxy + diz*qkxz;
      numtyp diqky = dix*qkxy + diy*qkyy + diz*qkyz;
      numtyp diqkz = dix*qkxz + diy*qkyz + diz*qkzz;
      numtyp dkqix = dkx*qixx + dky*qixy + dkz*qixz;
      numtyp dkqiy = dkx*qixy + dky*qiyy + dkz*qiyz;
      numtyp dkqiz = dkx*qixz + dky*qiyz + dkz*qizz;
      numtyp dkqirx = dkqiz*yr - dkqiy*zr;
      numtyp dkqiry = dkqix*zr - dkqiz*xr;
      numtyp dkqirz = dkqiy*xr - dkqix*yr;
      numtyp dqikx = diy*qkz - diz*qky + dky*qiz - dkz*qiy -
        (numtyp)2.0*(qixy*qkxz+qiyy*qkyz+qiyz*qkzz - qixz*qkxy-qiyz*qkyy-qizz*qkyz);
      numtyp dqiky = diz*qkx - dix*qkz + dkz*qix - dkx*qiz -
        (numtyp)2.0*(qixz*qkxx+qiyz*qkxy+qizz*qkxz - qixx*qkxz-qixy*qkyz-qixz*qkzz);
      numtyp dqikz = dix*qky - diy*qkx + dkx*qiy - dky*qix -
        (numtyp)2.0*(qixx*qkxy+qixy*qkyy+qixz*qkyz - qixy*qkxx-qiyy*qkxy-qiyz*qkxz);

      // get reciprocal distance terms for this interaction

      numtyp rinv = ucl_recip(r);
      numtyp r2inv = rinv*rinv;
      numtyp rr1 = felec * rinv;
      numtyp rr3 = rr1 * r2inv;
      numtyp rr5 = (numtyp)3.0 * rr3 * r2inv;
      numtyp rr7 = (numtyp)5.0 * rr5 * r2inv;
      numtyp rr9 = (numtyp)7.0 * rr7 * r2inv;
      numtyp rr11 = (numtyp)9.0 * rr9 * r2inv;

      // calculate the real space Ewald error function terms

      numtyp ralpha = aewald * r;
      numtyp exp2a = ucl_exp(-ralpha*ralpha);
      numtyp bn[6];
      bn[0] = ucl_erfc(ralpha) * rinv;

      numtyp alsq2 = (numtyp)2.0 * aewald*aewald;
      numtyp alsq2n = (numtyp)0.0;
      if (aewald > (numtyp)0.0) alsq2n = (numtyp)1.0 / (MY_PIS*aewald);

      int m;
      for (m = 1; m < 6; m++) {
        numtyp bfac = (numtyp) (m+m-1);
        alsq2n = alsq2 * alsq2n;
        bn[m] = (bfac*bn[m-1]+alsq2n*exp2a) * r2inv;
      }
      for (m = 0; m < 6; m++) bn[m] *= felec;

      numtyp term1,term2,term3;
      numtyp term4,term5,term6;

      term1 = ci*ck;
      term2 = ck*dir - ci*dkr + dik;
      term3 = ci*qkr + ck*qir - dir*dkr + (numtyp)2.0*(dkqi-diqk+qiqk);
      term4 = dir*qkr - dkr*qir - (numtyp)4.0*qik;
      term5 = qir*qkr;
      numtyp scalek = (numtyp)1.0 - factor_mpole;
      rr1 = bn[0] - scalek*rr1;
      rr3 = bn[1] - scalek*rr3;
      rr5 = bn[2] - scalek*rr5;
      rr7 = bn[3] - scalek*rr7;
      rr9 = bn[4] - scalek*rr9;
      rr11 = bn[5] - scalek*rr11;
      numtyp e = term1*rr1 + term2*rr3 + term3*rr5 + term4*rr7 + term5*rr9;

      // find standard multipole intermediates for force and torque

      numtyp de = term1*rr3 + term2*rr5 + term3*rr7 + term4*rr9 + term5*rr11;
      term1 = -ck*rr3 + dkr*rr5 - qkr*rr7;
      term2 = ci*rr3 + dir*rr5 + qir*rr7;
      term3 = (numtyp)2.0 * rr5;
      term4 = (numtyp)2.0 * (-ck*rr5+dkr*rr7-qkr*rr9);
      term5 = (numtyp)2.0 * (-ci*rr5-dir*rr7-qir*rr9);
      term6 = (numtyp)4.0 * rr7;

      energy += e;

      // compute the force components for this interaction

      numtyp frcx = de*xr + term1*dix + term2*dkx + term3*(diqkx-dkqix) +
        term4*qix + term5*qkx + term6*(qixk+qkxi);
      numtyp frcy = de*yr + term1*diy + term2*dky + term3*(diqky-dkqiy) +
        term4*qiy + term5*qky + term6*(qiyk+qkyi);
      numtyp frcz = de*zr + term1*diz + term2*dkz + term3*(diqkz-dkqiz) +
        term4*qiz + term5*qkz + term6*(qizk+qkzi);

      // compute the torque components for this interaction

      numtyp ttmix = -rr3*dikx + term1*dirx + term3*(dqikx+dkqirx) -
        term4*qirx - term6*(qikrx+qikx);
      numtyp ttmiy = -rr3*diky + term1*diry + term3*(dqiky+dkqiry) -
        term4*qiry - term6*(qikry+qiky);
      numtyp ttmiz = -rr3*dikz + term1*dirz + term3*(dqikz+dkqirz) -
        term4*qirz - term6*(qikrz+qikz);

      // increment force-based gradient and torque on first site

      f.x -= frcx;
      f.y -= frcy;
      f.z -= frcz;
      tq.x += ttmix;
      tq.y += ttmiy;
      tq.z += ttmiz;

      if (EVFLAG && vflag) {
        numtyp vxx = -xr * frcx;
        numtyp vxy = (numtyp)-0.5 * (yr*frcx+xr*frcy);
        numtyp vxz = (numtyp)-0.5 * (zr*frcx+xr*frcz);
        numtyp vyy = -yr * frcy;
        numtyp vyz = (numtyp)-0.5 * (zr*frcy+yr*frcz);
        numtyp vzz = -zr * frcz;

        virial[0] -= vxx;
        virial[1] -= vyy;
        virial[2] -= vzz;
        virial[3] -= vxy;
        virial[4] -= vxz;
        virial[5] -= vyz;
      }
    } // nbor

  } // ii<inum

  // accumulate tq
  store_answers_amoeba_tq(tq,ii,inum,tid,t_per_atom,offset,i,tep);

  // accumate force, energy and virial: use _acc if not the first kernel
  store_answers_q(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,
     offset,eflag,vflag,ans,engv);
}

/* ----------------------------------------------------------------------
  udirect2b = Ewald real direct field via list
  udirect2b computes the real space contribution of the permanent
   atomic multipole moments to the field via a neighbor list
------------------------------------------------------------------------- */

__kernel void k_amoeba_udirect2b(const __global numtyp4 *restrict x_,
                                 const __global numtyp4 *restrict extra,
                                 const __global numtyp4 *restrict coeff,
                                 const __global numtyp4 *restrict sp_amoeba,
                                 const __global int *dev_nbor,
                                 const __global int *dev_packed,
                                 const __global int *dev_short_nbor,
                                 __global acctyp4 *restrict fieldp,
                                 const int inum,  const int nall,
                                 const int nbor_pitch, const int t_per_atom,
                                 const numtyp aewald, const numtyp off2,
                                 const numtyp polar_dscale, const numtyp polar_uscale)
{
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_ufld();

  acctyp _fieldp[6];
  for (int l=0; l<6; l++) _fieldp[l]=(acctyp)0;

  const __global numtyp4* polar1 = &extra[0];
  const __global numtyp4* polar2 = &extra[nall];
  const __global numtyp4* polar3 = &extra[2*nall];

  if (ii<inum) {
    int numj, nbor, nbor_end;
    const __global int* nbor_mem=dev_packed;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor];
      nbor += n_stride;
      nbor_end = nbor+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    const numtyp4 pol3i = polar3[i];
    int itype  = pol3i.z;    // amtype[i];
    int igroup = pol3i.w;    // amgroup[i];

    numtyp pdi = coeff[itype].x;
    numtyp pti = coeff[itype].y;
    numtyp ddi = coeff[itype].z;

    numtyp aesq2 = (numtyp)2.0 * aewald*aewald;
    numtyp aesq2n = (numtyp)0.0;
    if (aewald > (numtyp)0.0) aesq2n = (numtyp)1.0 / (MY_PIS*aewald);

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int jextra=nbor_mem[nbor];
      int j = jextra & NEIGHMASK15;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp xr = jx.x - ix.x;
      numtyp yr = jx.y - ix.y;
      numtyp zr = jx.z - ix.z;
      numtyp r2 = xr*xr + yr*yr + zr*zr;

      numtyp r = ucl_sqrt(r2);
      numtyp rinv = ucl_rsqrt(r2);
      numtyp r2inv = rinv*rinv;
      numtyp rr1 = rinv;
      numtyp rr3 = rr1 * r2inv;
      numtyp rr5 = (numtyp)3.0 * rr3 * r2inv;
      numtyp rr7 = (numtyp)5.0 * rr5 * r2inv;

      const numtyp4 pol1j = polar1[j];
      numtyp ck  = pol1j.x;  // rpole[j][0];
      numtyp dkx = pol1j.y;  // rpole[j][1];
      numtyp dky = pol1j.z;  // rpole[j][2];
      numtyp dkz = pol1j.w;  // rpole[j][3];
      const numtyp4 pol2j = polar2[j];
      numtyp qkxx = pol2j.x; // rpole[j][4];
      numtyp qkxy = pol2j.y; // rpole[j][5];
      numtyp qkxz = pol2j.z; // rpole[j][6];
      numtyp qkyy = pol2j.w; // rpole[j][8];
      const numtyp4 pol3j = polar3[j];
      numtyp qkyz = pol3j.x; // rpole[j][9];
      numtyp qkzz = pol3j.y; // rpole[j][12];
      int jtype = pol3j.z; // amtype[j];
      int jgroup =  pol3j.w; // amgroup[j];

      numtyp factor_dscale, factor_pscale;
      const numtyp4 sp_pol = sp_amoeba[sbmask15(jextra)];
      if (igroup == jgroup) {
        factor_pscale = sp_pol.y; // sp_amoeba_piscale[sbmask15(jextra)];
        factor_dscale = polar_dscale;
      } else {
        factor_pscale = sp_pol.z; // sp_amoeba_pscale[sbmask15(jextra)];
        factor_dscale = (numtyp)1.0;
      }

      // intermediates involving moments and separation distance

      numtyp dkr = dkx*xr + dky*yr + dkz*zr;
      numtyp qkx = qkxx*xr + qkxy*yr + qkxz*zr;
      numtyp qky = qkxy*xr + qkyy*yr + qkyz*zr;
      numtyp qkz = qkxz*xr + qkyz*yr + qkzz*zr;
      numtyp qkr = qkx*xr + qky*yr + qkz*zr;

      // calculate the real space Ewald error function terms

      numtyp ralpha = aewald * r;
      numtyp exp2a = ucl_exp(-ralpha*ralpha);
      numtyp bn[4], bcn[3];
      bn[0] = ucl_erfc(ralpha) * rinv;

      numtyp aefac = aesq2n;
      for (int m = 1; m <= 3; m++) {
        numtyp bfac = (numtyp) (m+m-1);
        aefac = aesq2 * aefac;
        bn[m] = (bfac*bn[m-1]+aefac*exp2a) * r2inv;
      }

      // find the field components for Thole polarization damping

      numtyp scale3 = (numtyp)1.0;
      numtyp scale5 = (numtyp)1.0;
      numtyp scale7 = (numtyp)1.0;
      numtyp damp = pdi * coeff[jtype].x; // pdamp[jtype]
      if (damp != (numtyp)0.0) {
        numtyp pgamma = MIN(ddi,coeff[jtype].z); // dirdamp[jtype]
        if (pgamma != (numtyp)0.0) {
          numtyp tmp = r*ucl_recip(damp);
          damp = pgamma * ucl_sqrt(tmp*tmp*tmp);
          if (damp < (numtyp)50.0) {
            numtyp expdamp = ucl_exp(-damp) ;
            scale3 = (numtyp)1.0 - expdamp ;
            scale5 = (numtyp)1.0 - expdamp*((numtyp)1.0+(numtyp)0.5*damp);
            scale7 = (numtyp)1.0 - expdamp*((numtyp)1.0+(numtyp)0.65*damp + (numtyp)0.15*damp*damp);
          }
        } else {
          pgamma = MIN(pti,coeff[jtype].y); // thole[jtype]
          numtyp tmp = r*ucl_recip(damp);
          damp = pgamma * (tmp*tmp*tmp);
          if (damp < (numtyp)50.0) {
            numtyp expdamp = ucl_exp(-damp);
            scale3 = (numtyp)1.0 - expdamp;
            scale5 = (numtyp)1.0 - expdamp*((numtyp)1.0+damp);
            scale7 = (numtyp)1.0 - expdamp*((numtyp)1.0+damp + (numtyp)0.6*damp*damp);
          }
        }
      } else { // damp == 0: ???
      }

      numtyp scalek = factor_dscale;
      bcn[0] = bn[1] - ((numtyp)1.0-scalek*scale3)*rr3;
      bcn[1] = bn[2] - ((numtyp)1.0-scalek*scale5)*rr5;
      bcn[2] = bn[3] - ((numtyp)1.0-scalek*scale7)*rr7;

      numtyp fid[3];
      fid[0] = -xr*(bcn[0]*ck-bcn[1]*dkr+bcn[2]*qkr) - bcn[0]*dkx + (numtyp)2.0*bcn[1]*qkx;
      fid[1] = -yr*(bcn[0]*ck-bcn[1]*dkr+bcn[2]*qkr) - bcn[0]*dky + (numtyp)2.0*bcn[1]*qky;
      fid[2] = -zr*(bcn[0]*ck-bcn[1]*dkr+bcn[2]*qkr) - bcn[0]*dkz + (numtyp)2.0*bcn[1]*qkz;

      scalek = factor_pscale;
      bcn[0] = bn[1] - ((numtyp)1.0-scalek*scale3)*rr3;
      bcn[1] = bn[2] - ((numtyp)1.0-scalek*scale5)*rr5;
      bcn[2] = bn[3] - ((numtyp)1.0-scalek*scale7)*rr7;
      numtyp fip[3];
      fip[0] = -xr*(bcn[0]*ck-bcn[1]*dkr+bcn[2]*qkr) - bcn[0]*dkx + (numtyp)2.0*bcn[1]*qkx;
      fip[1] = -yr*(bcn[0]*ck-bcn[1]*dkr+bcn[2]*qkr) - bcn[0]*dky + (numtyp)2.0*bcn[1]*qky;
      fip[2] = -zr*(bcn[0]*ck-bcn[1]*dkr+bcn[2]*qkr) - bcn[0]*dkz + (numtyp)2.0*bcn[1]*qkz;

      _fieldp[0] += fid[0];
      _fieldp[1] += fid[1];
      _fieldp[2] += fid[2];
      _fieldp[3] += fip[0];
      _fieldp[4] += fip[1];
      _fieldp[5] += fip[2];
    }  // nbor

  } // ii<inum

  // accumulate field and fieldp

  store_answers_fieldp(_fieldp,ii,inum,tid,t_per_atom,offset,i,fieldp);
}

/* ----------------------------------------------------------------------
  umutual2b = Ewald real mutual field via list
   umutual2b computes the real space contribution of the induced
   atomic dipole moments to the field via a neighbor list
------------------------------------------------------------------------- */

__kernel void k_amoeba_umutual2b(const __global numtyp4 *restrict x_,
                                 const __global numtyp4 *restrict extra,
                                 const __global numtyp4 *restrict coeff,
                                 const __global numtyp4 *restrict sp_amoeba,
                                 const __global int *dev_nbor,
                                 const __global int *dev_packed,
                                 const __global int *dev_short_nbor,
                                 __global acctyp4 *restrict fieldp,
                                 const int inum,  const int nall,
                                 const int nbor_pitch, const int t_per_atom,
                                 const numtyp aewald, const numtyp off2,
                                 const numtyp polar_dscale, const numtyp polar_uscale)
{
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_ufld();

  acctyp _fieldp[6];
  for (int l=0; l<6; l++) _fieldp[l]=(acctyp)0;

  const __global numtyp4* polar3 = &extra[2*nall];
  const __global numtyp4* polar4 = &extra[3*nall];
  const __global numtyp4* polar5 = &extra[4*nall];

  if (ii<inum) {
    int numj, nbor, nbor_end;
    const __global int* nbor_mem=dev_packed;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor];
      nbor += n_stride;
      nbor_end = nbor+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    int itype,igroup;
    itype  = polar3[i].z; // amtype[i];
    igroup = polar3[i].w; // amgroup[i];

    numtyp pdi = coeff[itype].x;
    numtyp pti = coeff[itype].y;

    numtyp aesq2 = (numtyp)2.0 * aewald*aewald;
    numtyp aesq2n = (numtyp)0.0;
    if (aewald > (numtyp)0.0) aesq2n = (numtyp)1.0 / (MY_PIS*aewald);

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int jextra=nbor_mem[nbor];
      int j = jextra & NEIGHMASK15;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp xr = jx.x - ix.x;
      numtyp yr = jx.y - ix.y;
      numtyp zr = jx.z - ix.z;
      numtyp r2 = xr*xr + yr*yr + zr*zr;

      numtyp r = ucl_sqrt(r2);
      numtyp rinv = ucl_rsqrt(r2);
      numtyp r2inv = rinv*rinv;
      numtyp rr1 = rinv;
      numtyp rr3 = rr1 * r2inv;
      numtyp rr5 = (numtyp)3.0 * rr3 * r2inv;

      const numtyp4 pol3j = polar3[j];
      int jtype = pol3j.z; // amtype[j];
      int jgroup =  pol3j.w; // amgroup[j];
      const numtyp4 pol4j = polar4[j];
      numtyp ukx = pol4j.x;  // uind[j][0];
      numtyp uky = pol4j.y;  // uind[j][1];
      numtyp ukz = pol4j.z;  // uind[j][2];
      const numtyp4 pol5j = polar5[j];
      numtyp ukxp = pol5j.x; // uinp[j][0];
      numtyp ukyp = pol5j.y; // uinp[j][1];
      numtyp ukzp = pol5j.z; // uinp[j][2];

      numtyp factor_uscale;
      if (igroup == jgroup) factor_uscale = polar_uscale;
      else factor_uscale = (numtyp)1.0;

      // calculate the real space Ewald error function terms

      numtyp ralpha = aewald * r;
      numtyp exp2a = ucl_exp(-ralpha*ralpha);
      numtyp bn[4];
      bn[0] = ucl_erfc(ralpha) * rinv;

      numtyp aefac = aesq2n;
      for (int m = 1; m <= 3; m++) {
        numtyp bfac = (numtyp) (m+m-1);
        aefac = aesq2 * aefac;
        bn[m] = (bfac*bn[m-1]+aefac*exp2a) * r2inv;
      }

      // find terms needed later to compute mutual polarization
      // if (poltyp != DIRECT)
      numtyp scale3 = (numtyp)1.0;
      numtyp scale5 = (numtyp)1.0;
      numtyp damp = pdi * coeff[jtype].x; // pdamp[jtype]
      if (damp != (numtyp)0.0) {
        numtyp pgamma = MIN(pti,coeff[jtype].y); // thole[jtype]
        damp = pgamma * ucl_powr(r/damp,(numtyp)3.0);
        if (damp < (numtyp)50.0) {
          numtyp expdamp = ucl_exp(-damp);
          scale3 = (numtyp)1.0 - expdamp;
          scale5 = (numtyp)1.0 - expdamp*((numtyp)1.0+damp);
        }

      } else { // damp == 0: ???
      }

      numtyp scalek = factor_uscale;
      numtyp bcn[3];
      bcn[0] = bn[1] - ((numtyp)1.0-scalek*scale3)*rr3;
      bcn[1] = bn[2] - ((numtyp)1.0-scalek*scale5)*rr5;

      numtyp tdipdip[6];
      tdipdip[0] = -bcn[0] + bcn[1]*xr*xr;
      tdipdip[1] = bcn[1]*xr*yr;
      tdipdip[2] = bcn[1]*xr*zr;
      tdipdip[3] = -bcn[0] + bcn[1]*yr*yr;
      tdipdip[4] = bcn[1]*yr*zr;
      tdipdip[5] = -bcn[0] + bcn[1]*zr*zr;

      numtyp fid[3];
      fid[0] = tdipdip[0]*ukx + tdipdip[1]*uky + tdipdip[2]*ukz;
      fid[1] = tdipdip[1]*ukx + tdipdip[3]*uky + tdipdip[4]*ukz;
      fid[2] = tdipdip[2]*ukx + tdipdip[4]*uky + tdipdip[5]*ukz;

      numtyp fip[3];
      fip[0] = tdipdip[0]*ukxp + tdipdip[1]*ukyp + tdipdip[2]*ukzp;
      fip[1] = tdipdip[1]*ukxp + tdipdip[3]*ukyp + tdipdip[4]*ukzp;
      fip[2] = tdipdip[2]*ukxp + tdipdip[4]*ukyp + tdipdip[5]*ukzp;

      _fieldp[0] += fid[0];
      _fieldp[1] += fid[1];
      _fieldp[2] += fid[2];
      _fieldp[3] += fip[0];
      _fieldp[4] += fip[1];
      _fieldp[5] += fip[2];
    }  // nbor

  } // ii<inum

  // accumulate field and fieldp

  store_answers_fieldp(_fieldp,ii,inum,tid,t_per_atom,offset,i,fieldp);
}

/* ----------------------------------------------------------------------
   polar_real = real-space portion of induced dipole polarization
   adapted from Tinker epreal1d() routine
------------------------------------------------------------------------- */

__kernel void k_amoeba_polar(const __global numtyp4 *restrict x_,
                             const __global numtyp4 *restrict extra,
                             const __global numtyp4 *restrict coeff,
                             const __global numtyp4 *restrict sp_amoeba,
                             const __global int *dev_nbor,
                             const __global int *dev_packed,
                             const __global int *dev_short_nbor,
                             __global acctyp4 *restrict ans,
                             __global acctyp *restrict engv,
                             __global acctyp4 *restrict tep,
                             const int eflag, const int vflag, const int inum,
                             const int nall, const int nbor_pitch, const int t_per_atom,
                             const numtyp aewald, const numtyp felec,
                             const numtyp off2, const numtyp polar_dscale,
                             const numtyp polar_uscale)
{
  int tid, ii, offset, i;
  atom_info(t_per_atom,ii,tid,offset);

  int n_stride;
  local_allocate_store_charge();

  acctyp4 f;
  f.x=(acctyp)0; f.y=(acctyp)0; f.z=(acctyp)0;
  acctyp energy, e_coul, virial[6];
  if (EVFLAG) {
    energy=(acctyp)0;
    e_coul=(acctyp)0;
    for (int l=0; l<6; l++) virial[l]=(acctyp)0;
  }

  acctyp ufld[3];
  ufld[0] = (acctyp)0; ufld[1]=(acctyp)0; ufld[2]=(acctyp)0;
  acctyp dufld[6];
  for (int l=0; l<6; l++) dufld[l]=(acctyp)0;

  numtyp dix,diy,diz,qixx,qixy,qixz,qiyy,qiyz,qizz;
  
  const __global numtyp4* polar1 = &extra[0];
  const __global numtyp4* polar2 = &extra[nall];
  const __global numtyp4* polar3 = &extra[2*nall];
  const __global numtyp4* polar4 = &extra[3*nall];
  const __global numtyp4* polar5 = &extra[4*nall];

  if (ii<inum) {
    int itype,igroup;
    numtyp ci,uix,uiy,uiz,uixp,uiyp,uizp;

    int numj, nbor, nbor_end;
    const __global int* nbor_mem=dev_packed;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];

    // recalculate numj and nbor_end for use of the short nbor list
    if (dev_packed==dev_nbor) {
      numj = dev_short_nbor[nbor];
      nbor += n_stride;
      nbor_end = nbor+fast_mul(numj,n_stride);
      nbor_mem = dev_short_nbor;
    }

    const numtyp4 pol1i = polar1[i];
    ci  = pol1i.x;    // rpole[i][0];
    dix = pol1i.y;    // rpole[i][1];
    diy = pol1i.z;    // rpole[i][2];
    diz = pol1i.w;    // rpole[i][3];
    const numtyp4 pol2i = polar2[i];
    qixx = pol2i.x;   // rpole[i][4];
    qixy = pol2i.y;   // rpole[i][5];
    qixz = pol2i.z;   // rpole[i][6];
    qiyy = pol2i.w;   // rpole[i][8];
    const numtyp4 pol3i = polar3[i];
    qiyz = pol3i.x;   // rpole[i][9];
    qizz = pol3i.y;   // rpole[i][12];
    itype  = pol3i.z;    // amtype[i];
    igroup = pol3i.w;    // amgroup[i];
    const numtyp4 pol4i = polar4[i];
    uix = pol4i.x;    // uind[i][0];
    uiy = pol4i.y;    // uind[i][1];
    uiz = pol4i.z;    // uind[i][2];
    const numtyp4 pol5i = polar5[i];
    uixp = pol5i.x;   // uinp[i][0];
    uiyp = pol5i.y;   // uinp[i][1];
    uizp = pol5i.z;   // uinp[i][2];

    numtyp pdi = coeff[itype].x;
    numtyp pti = coeff[itype].y;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int jextra=nbor_mem[nbor];
      int j = jextra & NEIGHMASK15;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp xr = jx.x - ix.x;
      numtyp yr = jx.y - ix.y;
      numtyp zr = jx.z - ix.z;
      numtyp r2 = xr*xr + yr*yr + zr*zr;
      numtyp r = ucl_sqrt(r2);

      const numtyp4 pol1j = polar1[j];
      numtyp ck = pol1j.x;   // rpole[j][0];
      numtyp dkx = pol1j.y;  // rpole[j][1];
      numtyp dky = pol1j.z;  // rpole[j][2];
      numtyp dkz = pol1j.w;  // rpole[j][3];
      const numtyp4 pol2j = polar2[j];
      numtyp qkxx = pol2j.x; // rpole[j][4];
      numtyp qkxy = pol2j.y; // rpole[j][5];
      numtyp qkxz = pol2j.z; // rpole[j][6];
      numtyp qkyy = pol2j.w; // rpole[j][8];
      const numtyp4 pol3j = polar3[j];
      numtyp qkyz = pol3j.x; // rpole[j][9];
      numtyp qkzz = pol3j.y; // rpole[j][12];
      int jtype =   pol3j.z; // amtype[j];
      int jgroup =  pol3j.w; // amgroup[j];
      const numtyp4 pol4j = polar4[j];
      numtyp ukx = pol4j.x;  // uind[j][0];
      numtyp uky = pol4j.y;  // uind[j][1];
      numtyp ukz = pol4j.z;  // uind[j][2];
      const numtyp4 pol5j = polar5[j];
      numtyp ukxp = pol5j.x; // uinp[j][0];
      numtyp ukyp = pol5j.y; // uinp[j][1];
      numtyp ukzp = pol5j.z; // uinp[j][2];

      numtyp factor_dscale, factor_pscale, factor_uscale;
      const numtyp4 sp_pol = sp_amoeba[sbmask15(jextra)];
      if (igroup == jgroup) {
        factor_pscale = sp_pol.y; // sp_amoeba_piscale[sbmask15(jextra)];
        factor_dscale = polar_dscale;
        factor_uscale = polar_uscale;
      } else {
        factor_pscale = sp_pol.z; // sp_amoeba_pscale[sbmask15(jextra)];
        factor_dscale = factor_uscale = (numtyp)1.0;
      }

      // intermediates involving moments and separation distance

      numtyp dir = dix*xr + diy*yr + diz*zr;
      numtyp qix = qixx*xr + qixy*yr + qixz*zr;
      numtyp qiy = qixy*xr + qiyy*yr + qiyz*zr;
      numtyp qiz = qixz*xr + qiyz*yr + qizz*zr;
      numtyp qir = qix*xr + qiy*yr + qiz*zr;
      numtyp dkr = dkx*xr + dky*yr + dkz*zr;
      numtyp qkx = qkxx*xr + qkxy*yr + qkxz*zr;
      numtyp qky = qkxy*xr + qkyy*yr + qkyz*zr;
      numtyp qkz = qkxz*xr + qkyz*yr + qkzz*zr;
      numtyp qkr = qkx*xr + qky*yr + qkz*zr;
      numtyp uir = uix*xr + uiy*yr + uiz*zr;
      numtyp ukr = ukx*xr + uky*yr + ukz*zr;
      numtyp ukrp = ukxp*xr + ukyp*yr + ukzp*zr;

      // get reciprocal distance terms for this interaction

      numtyp rinv = ucl_recip(r);
      numtyp r2inv = rinv*rinv;
      numtyp rr1 = felec * rinv;
      numtyp rr3 = rr1 * r2inv;
      numtyp rr5 = (numtyp)3.0 * rr3 * r2inv;
      numtyp rr7 = (numtyp)5.0 * rr5 * r2inv;
      numtyp rr9 = (numtyp)7.0 * rr7 * r2inv;

      // calculate the real space Ewald error function terms

      int k,m;
      numtyp psc3,psc5,psc7;
      numtyp dsc3,dsc5,dsc7;
      numtyp usc3,usc5;
      numtyp psr3,psr5,psr7;
      numtyp dsr3,dsr5,dsr7;
      numtyp usr5;
      numtyp term1,term2,term3;
      numtyp term4,term5;
      numtyp term6,term7;
      numtyp rc3[3],rc5[3],rc7[3];
      numtyp prc3[3],prc5[3],prc7[3];
      numtyp drc3[3],drc5[3],drc7[3];
      numtyp urc3[3],urc5[3];
    
      numtyp ralpha = aewald * r;
      numtyp exp2a = ucl_exp(-ralpha*ralpha);
      numtyp bn[5];
      bn[0] = ucl_erfc(ralpha) * rinv;

      numtyp alsq2 = (numtyp)2.0 * aewald*aewald;
      numtyp alsq2n = (numtyp)0.0;
      if (aewald > (numtyp)0.0) alsq2n = (numtyp)1.0 / (MY_PIS*aewald);

      for (m = 1; m <= 4; m++) {
        numtyp bfac = (numtyp) (m+m-1);
        alsq2n = alsq2 * alsq2n;
        bn[m] = (bfac*bn[m-1]+alsq2n*exp2a) * r2inv;
      }
      for (m = 0; m < 5; m++) bn[m] *= felec;

      // apply Thole polarization damping to scale factors

      numtyp sc3 = (numtyp)1.0;
      numtyp sc5 = (numtyp)1.0;
      numtyp sc7 = (numtyp)1.0;
      for (k = 0; k < 3; k++) {
        rc3[k] = (numtyp)0.0;
        rc5[k] = (numtyp)0.0;
        rc7[k] = (numtyp)0.0;
      }

      // apply Thole polarization damping to scale factors

      numtyp damp = pdi * coeff[jtype].x; // pdamp[jtype]
      if (damp != (numtyp)0.0) {
        numtyp pgamma = MIN(pti,coeff[jtype].y); // thole[jtype]
        numtyp tmp = r*ucl_recip(damp);
        damp = pgamma * (tmp*tmp*tmp);
        if (damp < (numtyp)50.0) {
          numtyp expdamp = ucl_exp(-damp);
          sc3 = (numtyp)1.0 - expdamp;
          sc5 = (numtyp)1.0 - ((numtyp)1.0+damp)*expdamp;
          sc7 = (numtyp)1.0 - ((numtyp)1.0+damp+(numtyp)0.6*damp*damp) * expdamp;
          numtyp temp3 = (numtyp)3.0 * damp * expdamp * r2inv;
          numtyp temp5 = damp;
          numtyp temp7 = (numtyp)-0.2 + (numtyp)0.6*damp;
          rc3[0] = xr * temp3;
          rc3[1] = yr * temp3;
          rc3[2] = zr * temp3;
          rc5[0] = rc3[0] * temp5;
          rc5[1] = rc3[1] * temp5;
          rc5[2] = rc3[2] * temp5;
          rc7[0] = rc5[0] * temp7;
          rc7[1] = rc5[1] * temp7;
          rc7[2] = rc5[2] * temp7;
        }

        psc3 = (numtyp)1.0 - sc3*factor_pscale;
        psc5 = (numtyp)1.0 - sc5*factor_pscale;
        psc7 = (numtyp)1.0 - sc7*factor_pscale;
        dsc3 = (numtyp)1.0 - sc3*factor_dscale;
        dsc5 = (numtyp)1.0 - sc5*factor_dscale;
        dsc7 = (numtyp)1.0 - sc7*factor_dscale;
        usc3 = (numtyp)1.0 - sc3*factor_uscale;
        usc5 = (numtyp)1.0 - sc5*factor_uscale;
        psr3 = bn[1] - psc3*rr3;
        psr5 = bn[2] - psc5*rr5;
        psr7 = bn[3] - psc7*rr7;
        dsr3 = bn[1] - dsc3*rr3;
        dsr5 = bn[2] - dsc5*rr5;
        dsr7 = bn[3] - dsc7*rr7;
        usr5 = bn[2] - usc5*rr5;
        for (k = 0; k < 3; k++) {
          prc3[k] = rc3[k] * factor_pscale;
          prc5[k] = rc5[k] * factor_pscale;
          prc7[k] = rc7[k] * factor_pscale;
          drc3[k] = rc3[k] * factor_dscale;
          drc5[k] = rc5[k] * factor_dscale;
          drc7[k] = rc7[k] * factor_dscale;
          urc3[k] = rc3[k] * factor_uscale;
          urc5[k] = rc5[k] * factor_uscale;
        }
      } else { // damp == 0: ???
      }

      // get the induced dipole field used for dipole torques

      numtyp tix3 = psr3*ukx + dsr3*ukxp;
      numtyp tiy3 = psr3*uky + dsr3*ukyp;
      numtyp tiz3 = psr3*ukz + dsr3*ukzp;
      numtyp tuir = -psr5*ukr - dsr5*ukrp;

      ufld[0] += tix3 + xr*tuir;
      ufld[1] += tiy3 + yr*tuir;
      ufld[2] += tiz3 + zr*tuir;

      // get induced dipole field gradient used for quadrupole torques

      numtyp tix5 = (numtyp)2.0 * (psr5*ukx+dsr5*ukxp);
      numtyp tiy5 = (numtyp)2.0 * (psr5*uky+dsr5*ukyp);
      numtyp tiz5 = (numtyp)2.0 * (psr5*ukz+dsr5*ukzp);
      tuir = -psr7*ukr - dsr7*ukrp;

      dufld[0] += xr*tix5 + xr*xr*tuir;
      dufld[1] += xr*tiy5 + yr*tix5 + (numtyp)2.0*xr*yr*tuir;
      dufld[2] += yr*tiy5 + yr*yr*tuir;
      dufld[3] += xr*tiz5 + zr*tix5 + (numtyp)2.0*xr*zr*tuir;
      dufld[4] += yr*tiz5 + zr*tiy5 + (numtyp)2.0*yr*zr*tuir;
      dufld[5] += zr*tiz5 + zr*zr*tuir;

      // get the dEd/dR terms used for direct polarization force

      term1 = bn[2] - dsc3*rr5;
      term2 = bn[3] - dsc5*rr7;
      term3 = -dsr3 + term1*xr*xr - rr3*xr*drc3[0];
      term4 = rr3*drc3[0] - term1*xr - dsr5*xr;
      term5 = term2*xr*xr - dsr5 - rr5*xr*drc5[0];
      term6 = (bn[4]-dsc7*rr9)*xr*xr - bn[3] - rr7*xr*drc7[0];
      term7 = rr5*drc5[0] - (numtyp)2.0*bn[3]*xr + (dsc5+(numtyp)1.5*dsc7)*rr7*xr;
      numtyp tixx = ci*term3 + dix*term4 + dir*term5 +
        (numtyp)2.0*dsr5*qixx + (qiy*yr+qiz*zr)*dsc7*rr7 + (numtyp)2.0*qix*term7 + qir*term6;
      numtyp tkxx = ck*term3 - dkx*term4 - dkr*term5 +
        (numtyp)2.0*dsr5*qkxx + (qky*yr+qkz*zr)*dsc7*rr7 + (numtyp)2.0*qkx*term7 + qkr*term6;

      term3 = -dsr3 + term1*yr*yr - rr3*yr*drc3[1];
      term4 = rr3*drc3[1] - term1*yr - dsr5*yr;
      term5 = term2*yr*yr - dsr5 - rr5*yr*drc5[1];
      term6 = (bn[4]-dsc7*rr9)*yr*yr - bn[3] - rr7*yr*drc7[1];
      term7 = rr5*drc5[1] - (numtyp)2.0*bn[3]*yr + (dsc5+(numtyp)1.5*dsc7)*rr7*yr;
      numtyp tiyy = ci*term3 + diy*term4 + dir*term5 +
        (numtyp)2.0*dsr5*qiyy + (qix*xr+qiz*zr)*dsc7*rr7 + (numtyp)2.0*qiy*term7 + qir*term6;
      numtyp tkyy = ck*term3 - dky*term4 - dkr*term5 +
        (numtyp)2.0*dsr5*qkyy + (qkx*xr+qkz*zr)*dsc7*rr7 + (numtyp)2.0*qky*term7 + qkr*term6;

      term3 = -dsr3 + term1*zr*zr - rr3*zr*drc3[2];
      term4 = rr3*drc3[2] - term1*zr - dsr5*zr;
      term5 = term2*zr*zr - dsr5 - rr5*zr*drc5[2];
      term6 = (bn[4]-dsc7*rr9)*zr*zr - bn[3] - rr7*zr*drc7[2];
      term7 = rr5*drc5[2] - (numtyp)2.0*bn[3]*zr + (dsc5+(numtyp)1.5*dsc7)*rr7*zr;
      numtyp tizz = ci*term3 + diz*term4 + dir*term5 +
        (numtyp)2.0*dsr5*qizz + (qix*xr+qiy*yr)*dsc7*rr7 + (numtyp)2.0*qiz*term7 + qir*term6;
      numtyp tkzz = ck*term3 - dkz*term4 - dkr*term5 +
        (numtyp)2.0*dsr5*qkzz + (qkx*xr+qky*yr)*dsc7*rr7 + (numtyp)2.0*qkz*term7 + qkr*term6;

      term3 = term1*xr*yr - rr3*yr*drc3[0];
      term4 = rr3*drc3[0] - term1*xr;
      term5 = term2*xr*yr - rr5*yr*drc5[0];
      term6 = (bn[4]-dsc7*rr9)*xr*yr - rr7*yr*drc7[0];
      term7 = rr5*drc5[0] - term2*xr;
      numtyp tixy = ci*term3 - dsr5*dix*yr + diy*term4 + dir*term5 +
        (numtyp)2.0*dsr5*qixy - (numtyp)2.0*dsr7*yr*qix + (numtyp)2.0*qiy*term7 + qir*term6;
      numtyp tkxy = ck*term3 + dsr5*dkx*yr - dky*term4 - dkr*term5 +
        (numtyp)2.0*dsr5*qkxy - (numtyp)2.0*dsr7*yr*qkx +(numtyp) 2.0*qky*term7 + qkr*term6;

      term3 = term1*xr*zr - rr3*zr*drc3[0];
      term5 = term2*xr*zr - rr5*zr*drc5[0];
      term6 = (bn[4]-dsc7*rr9)*xr*zr - rr7*zr*drc7[0];
      numtyp tixz = ci*term3 - dsr5*dix*zr + diz*term4 + dir*term5 +
        (numtyp)2.0*dsr5*qixz - (numtyp)2.0*dsr7*zr*qix + (numtyp)2.0*qiz*term7 + qir*term6;
      numtyp tkxz = ck*term3 + dsr5*dkx*zr - dkz*term4 - dkr*term5 +
        (numtyp)2.0*dsr5*qkxz - (numtyp)2.0*dsr7*zr*qkx + (numtyp)2.0*qkz*term7 + qkr*term6;

      term3 = term1*yr*zr - rr3*zr*drc3[1];
      term4 = rr3*drc3[1] - term1*yr;
      term5 = term2*yr*zr - rr5*zr*drc5[1];
      term6 = (bn[4]-dsc7*rr9)*yr*zr - rr7*zr*drc7[1];
      term7 = rr5*drc5[1] - term2*yr;
      numtyp tiyz = ci*term3 - dsr5*diy*zr + diz*term4 + dir*term5 +
        (numtyp)2.0*dsr5*qiyz - (numtyp)2.0*dsr7*zr*qiy + (numtyp)2.0*qiz*term7 + qir*term6;
      numtyp tkyz = ck*term3 + dsr5*dky*zr - dkz*term4 - dkr*term5 +
        (numtyp)2.0*dsr5*qkyz - (numtyp)2.0*dsr7*zr*qky + (numtyp)2.0*qkz*term7 + qkr*term6;

      numtyp depx = tixx*ukxp + tixy*ukyp + tixz*ukzp - tkxx*uixp - tkxy*uiyp - tkxz*uizp;
      numtyp depy = tixy*ukxp + tiyy*ukyp + tiyz*ukzp - tkxy*uixp - tkyy*uiyp - tkyz*uizp;
      numtyp depz = tixz*ukxp + tiyz*ukyp + tizz*ukzp - tkxz*uixp - tkyz*uiyp - tkzz*uizp;

      numtyp frcx = depx;
      numtyp frcy = depy;
      numtyp frcz = depz;

      // get the dEp/dR terms used for direct polarization force

      // tixx and tkxx
      term1 = bn[2] - psc3*rr5;
      term2 = bn[3] - psc5*rr7;
      term3 = -psr3 + term1*xr*xr - rr3*xr*prc3[0];
      term4 = rr3*prc3[0] - term1*xr - psr5*xr;
      term5 = term2*xr*xr - psr5 - rr5*xr*prc5[0];
      term6 = (bn[4]-psc7*rr9)*xr*xr - bn[3] - rr7*xr*prc7[0];
      term7 = rr5*prc5[0] - (numtyp)2.0*bn[3]*xr + (psc5+(numtyp)1.5*psc7)*rr7*xr;
      tixx = ci*term3 + dix*term4 + dir*term5 +
        (numtyp)2.0*psr5*qixx + (qiy*yr+qiz*zr)*psc7*rr7 + (numtyp)2.0*qix*term7 + qir*term6;
      tkxx = ck*term3 - dkx*term4 - dkr*term5 +
        (numtyp)2.0*psr5*qkxx + (qky*yr+qkz*zr)*psc7*rr7 + (numtyp)2.0*qkx*term7 + qkr*term6;

      // tiyy and tkyy
      term3 = -psr3 + term1*yr*yr - rr3*yr*prc3[1];
      term4 = rr3*prc3[1] - term1*yr - psr5*yr;
      term5 = term2*yr*yr - psr5 - rr5*yr*prc5[1];
      term6 = (bn[4]-psc7*rr9)*yr*yr - bn[3] - rr7*yr*prc7[1];
      term7 = rr5*prc5[1] - (numtyp)2.0*bn[3]*yr + (psc5+(numtyp)1.5*psc7)*rr7*yr;
      tiyy = ci*term3 + diy*term4 + dir*term5 +
        (numtyp)2.0*psr5*qiyy + (qix*xr+qiz*zr)*psc7*rr7 + (numtyp)2.0*qiy*term7 + qir*term6;
      tkyy = ck*term3 - dky*term4 - dkr*term5 +
        (numtyp)2.0*psr5*qkyy + (qkx*xr+qkz*zr)*psc7*rr7 + (numtyp)2.0*qky*term7 + qkr*term6;

      // tizz and tkzz
      term3 = -psr3 + term1*zr*zr - rr3*zr*prc3[2];
      term4 = rr3*prc3[2] - term1*zr - psr5*zr;
      term5 = term2*zr*zr - psr5 - rr5*zr*prc5[2];
      term6 = (bn[4]-psc7*rr9)*zr*zr - bn[3] - rr7*zr*prc7[2];
      term7 = rr5*prc5[2] - (numtyp)2.0*bn[3]*zr + (psc5+(numtyp)1.5*psc7)*rr7*zr;
      tizz = ci*term3 + diz*term4 + dir*term5 +
        (numtyp)2.0*psr5*qizz + (qix*xr+qiy*yr)*psc7*rr7 + (numtyp)2.0*qiz*term7 + qir*term6;
      tkzz = ck*term3 - dkz*term4 - dkr*term5 +
        (numtyp)2.0*psr5*qkzz + (qkx*xr+qky*yr)*psc7*rr7 + (numtyp)2.0*qkz*term7 + qkr*term6;

      // tixy and tkxy
      term3 = term1*xr*yr - rr3*yr*prc3[0];
      term4 = rr3*prc3[0] - term1*xr;
      term5 = term2*xr*yr - rr5*yr*prc5[0];
      term6 = (bn[4]-psc7*rr9)*xr*yr - rr7*yr*prc7[0];
      term7 = rr5*prc5[0] - term2*xr;
      tixy = ci*term3 - psr5*dix*yr + diy*term4 + dir*term5 +
        (numtyp)2.0*psr5*qixy - (numtyp)2.0*psr7*yr*qix + (numtyp)2.0*qiy*term7 + qir*term6;
      tkxy = ck*term3 + psr5*dkx*yr - dky*term4 - dkr*term5 +
        (numtyp)2.0*psr5*qkxy - (numtyp)2.0*psr7*yr*qkx + (numtyp)2.0*qky*term7 + qkr*term6;

      // tixz and tkxz
      term3 = term1*xr*zr - rr3*zr*prc3[0];
      term5 = term2*xr*zr - rr5*zr*prc5[0];
      term6 = (bn[4]-psc7*rr9)*xr*zr - rr7*zr*prc7[0];
      tixz = ci*term3 - psr5*dix*zr + diz*term4 + dir*term5 +
        (numtyp)2.0*psr5*qixz - (numtyp)2.0*psr7*zr*qix + (numtyp)2.0*qiz*term7 + qir*term6;
      tkxz = ck*term3 + psr5*dkx*zr - dkz*term4 - dkr*term5 +
        (numtyp)2.0*psr5*qkxz - (numtyp)2.0*psr7*zr*qkx + (numtyp)2.0*qkz*term7 + qkr*term6;

      // tiyz and tkyz
      term3 = term1*yr*zr - rr3*zr*prc3[1];
      term4 = rr3*prc3[1] - term1*yr;
      term5 = term2*yr*zr - rr5*zr*prc5[1];
      term6 = (bn[4]-psc7*rr9)*yr*zr - rr7*zr*prc7[1];
      term7 = rr5*prc5[1] - term2*yr;
      tiyz = ci*term3 - psr5*diy*zr + diz*term4 + dir*term5 +
        (numtyp)2.0*psr5*qiyz - (numtyp)2.0*psr7*zr*qiy + (numtyp)2.0*qiz*term7 + qir*term6;
      tkyz = ck*term3 + psr5*dky*zr - dkz*term4 - dkr*term5 +
        (numtyp)2.0*psr5*qkyz - (numtyp)2.0*psr7*zr*qky + (numtyp)2.0*qkz*term7 + qkr*term6;

      depx = tixx*ukx + tixy*uky + tixz*ukz - tkxx*uix - tkxy*uiy - tkxz*uiz;
      depy = tixy*ukx + tiyy*uky + tiyz*ukz - tkxy*uix - tkyy*uiy - tkyz*uiz;
      depz = tixz*ukx + tiyz*uky + tizz*ukz - tkxz*uix - tkyz*uiy - tkzz*uiz;

      frcx = frcx + depx;
      frcy = frcy + depy;
      frcz = frcz + depz;

      // get the dtau/dr terms used for mutual polarization force
      // poltyp == MUTUAL  && amoeba

      term1 = bn[2] - usc3*rr5;
      term2 = bn[3] - usc5*rr7;
      term3 = usr5 + term1;
      term4 = rr3 * factor_uscale;
      term5 = -xr*term3 + rc3[0]*term4;
      term6 = -usr5 + xr*xr*term2 - rr5*xr*urc5[0];
      tixx = uix*term5 + uir*term6;
      tkxx = ukx*term5 + ukr*term6;

      term5 = -yr*term3 + rc3[1]*term4;
      term6 = -usr5 + yr*yr*term2 - rr5*yr*urc5[1];
      tiyy = uiy*term5 + uir*term6;
      tkyy = uky*term5 + ukr*term6;

      term5 = -zr*term3 + rc3[2]*term4;
      term6 = -usr5 + zr*zr*term2 - rr5*zr*urc5[2];
      tizz = uiz*term5 + uir*term6;
      tkzz = ukz*term5 + ukr*term6;

      term4 = -usr5 * yr;
      term5 = -xr*term1 + rr3*urc3[0];
      term6 = xr*yr*term2 - rr5*yr*urc5[0];
      tixy = uix*term4 + uiy*term5 + uir*term6;
      tkxy = ukx*term4 + uky*term5 + ukr*term6;

      term4 = -usr5 * zr;
      term6 = xr*zr*term2 - rr5*zr*urc5[0];
      tixz = uix*term4 + uiz*term5 + uir*term6;
      tkxz = ukx*term4 + ukz*term5 + ukr*term6;

      term5 = -yr*term1 + rr3*urc3[1];
      term6 = yr*zr*term2 - rr5*zr*urc5[1];
      tiyz = uiy*term4 + uiz*term5 + uir*term6;
      tkyz = uky*term4 + ukz*term5 + ukr*term6;

      depx = tixx*ukxp + tixy*ukyp + tixz*ukzp
        + tkxx*uixp + tkxy*uiyp + tkxz*uizp;
      depy = tixy*ukxp + tiyy*ukyp + tiyz*ukzp
        + tkxy*uixp + tkyy*uiyp + tkyz*uizp;
      depz = tixz*ukxp + tiyz*ukyp + tizz*ukzp
        + tkxz*uixp + tkyz*uiyp + tkzz*uizp;

      frcx = frcx + depx;
      frcy = frcy + depy;
      frcz = frcz + depz;

      f.x += frcx;
      f.y += frcy;
      f.z += frcz;

      if (EVFLAG && vflag) {
        numtyp vxx = xr * frcx;
        numtyp vxy = (numtyp)0.5 * (yr*frcx+xr*frcy);
        numtyp vxz = (numtyp)0.5 * (zr*frcx+xr*frcz);
        numtyp vyy = yr * frcy;
        numtyp vyz = (numtyp)0.5 * (zr*frcy+yr*frcz);
        numtyp vzz = zr * frcz;

        virial[0] -= vxx;
        virial[1] -= vyy;
        virial[2] -= vzz;
        virial[3] -= vxy;
        virial[4] -= vxz;
        virial[5] -= vyz;
      }
    } // nbor

  } // ii<inum

  // accumulate ufld and dufld to compute tep
  store_answers_tep(ufld,dufld,ii,inum,tid,t_per_atom,offset,i,tep);

  // accumate force, energy and virial
  store_answers_acc(f,energy,e_coul,virial,ii,inum,tid,t_per_atom,
     offset,eflag,vflag,ans,engv,NUM_BLOCKS_X);
}

/* ----------------------------------------------------------------------
   fphi_uind = induced potential from grid
   fphi_uind extracts the induced dipole potential from the particle mesh Ewald grid
------------------------------------------------------------------------- */

__kernel void k_amoeba_fphi_uind(const __global numtyp4 *restrict thetai1,
                          const __global numtyp4 *restrict thetai2,
                          const __global numtyp4 *restrict thetai3,
                          const __global int *restrict igrid,
                          const __global numtyp2 *restrict grid,
                          __global acctyp *restrict fdip_phi1,
                          __global acctyp *restrict fdip_phi2,
                          __global acctyp *restrict fdip_sum_phi,
                          const int bsorder, const int inum,
                          const int nzlo_out, const int nylo_out,
                          const int nxlo_out, const int ngridxy,
                          const int ngridx)
{
  int tid=THREAD_ID_X;
  int ii=tid+BLOCK_ID_X*BLOCK_SIZE_X;

  if (ii<inum) {

    const int nlpts = (bsorder-1) / 2;
    
    int istart = fast_mul(ii,4);
    const int igridx = igrid[istart];
    const int igridy = igrid[istart+1];
    const int igridz = igrid[istart+2];
    
    // now istart is used to index thetai1, thetai2 and thetai3
    istart = fast_mul(ii,bsorder);

    // extract the permanent multipole field at each site

    numtyp tuv100_1 = (numtyp)0.0;
    numtyp tuv010_1 = (numtyp)0.0;
    numtyp tuv001_1 = (numtyp)0.0;
    numtyp tuv200_1 = (numtyp)0.0;
    numtyp tuv020_1 = (numtyp)0.0;
    numtyp tuv002_1 = (numtyp)0.0;
    numtyp tuv110_1 = (numtyp)0.0;
    numtyp tuv101_1 = (numtyp)0.0;
    numtyp tuv011_1 = (numtyp)0.0;
    numtyp tuv100_2 = (numtyp)0.0;
    numtyp tuv010_2 = (numtyp)0.0;
    numtyp tuv001_2 = (numtyp)0.0;
    numtyp tuv200_2 = (numtyp)0.0;
    numtyp tuv020_2 = (numtyp)0.0;
    numtyp tuv002_2 = (numtyp)0.0;
    numtyp tuv110_2 = (numtyp)0.0;
    numtyp tuv101_2 = (numtyp)0.0;
    numtyp tuv011_2 = (numtyp)0.0;
    numtyp tuv000 = (numtyp)0.0;
    numtyp tuv001 = (numtyp)0.0;
    numtyp tuv010 = (numtyp)0.0;
    numtyp tuv100 = (numtyp)0.0;
    numtyp tuv200 = (numtyp)0.0;
    numtyp tuv020 = (numtyp)0.0;
    numtyp tuv002 = (numtyp)0.0;
    numtyp tuv110 = (numtyp)0.0;
    numtyp tuv101 = (numtyp)0.0;
    numtyp tuv011 = (numtyp)0.0;
    numtyp tuv300 = (numtyp)0.0;
    numtyp tuv030 = (numtyp)0.0;
    numtyp tuv003 = (numtyp)0.0;
    numtyp tuv210 = (numtyp)0.0;
    numtyp tuv201 = (numtyp)0.0;
    numtyp tuv120 = (numtyp)0.0;
    numtyp tuv021 = (numtyp)0.0;
    numtyp tuv102 = (numtyp)0.0;
    numtyp tuv012 = (numtyp)0.0;
    numtyp tuv111 = (numtyp)0.0;

    int k = (igridz - nzlo_out) - nlpts;
    for (int kb = 0; kb < bsorder; kb++) {
      const int mz = fast_mul(k, ngridxy);
      const int i3 = istart + kb;
      const numtyp4 tha3 = thetai3[i3];
      const numtyp v0 = tha3.x; // thetai3[m][kb][0];
      const numtyp v1 = tha3.y; // thetai3[m][kb][1];
      const numtyp v2 = tha3.z; // thetai3[m][kb][2];
      const numtyp v3 = tha3.w; // thetai3[m][kb][3];
      numtyp tu00_1 = (numtyp)0.0;
      numtyp tu01_1 = (numtyp)0.0;
      numtyp tu10_1 = (numtyp)0.0;
      numtyp tu20_1 = (numtyp)0.0;
      numtyp tu11_1 = (numtyp)0.0;
      numtyp tu02_1 = (numtyp)0.0;
      numtyp tu00_2 = (numtyp)0.0;
      numtyp tu01_2 = (numtyp)0.0;
      numtyp tu10_2 = (numtyp)0.0;
      numtyp tu20_2 = (numtyp)0.0;
      numtyp tu11_2 = (numtyp)0.0;
      numtyp tu02_2 = (numtyp)0.0;
      numtyp tu00 = (numtyp)0.0;
      numtyp tu10 = (numtyp)0.0;
      numtyp tu01 = (numtyp)0.0;
      numtyp tu20 = (numtyp)0.0;
      numtyp tu11 = (numtyp)0.0;
      numtyp tu02 = (numtyp)0.0;
      numtyp tu30 = (numtyp)0.0;
      numtyp tu21 = (numtyp)0.0;
      numtyp tu12 = (numtyp)0.0;
      numtyp tu03 = (numtyp)0.0;

      int j = (igridy - nylo_out) - nlpts;
      for (int jb = 0; jb < bsorder; jb++) {
        const int my = mz + fast_mul(j, ngridx);
        const int i2 = istart + jb;
        const numtyp4 tha2 = thetai2[i2];
        const numtyp u0 = tha2.x; // thetai2[m][jb][0];
        const numtyp u1 = tha2.y; // thetai2[m][jb][1];
        const numtyp u2 = tha2.z; // thetai2[m][jb][2];
        const numtyp u3 = tha2.w; // thetai2[m][jb][3];
        numtyp t0_1 = (numtyp)0.0;
        numtyp t1_1 = (numtyp)0.0;
        numtyp t2_1 = (numtyp)0.0;
        numtyp t0_2 = (numtyp)0.0;
        numtyp t1_2 = (numtyp)0.0;
        numtyp t2_2 = (numtyp)0.0;
        numtyp t3 = (numtyp)0.0;

        int i = (igridx - nxlo_out) - nlpts;
        for (int ib = 0; ib < bsorder; ib++) {
          const int i1 = istart + ib;
          const numtyp4 tha1 = thetai1[i1];
          const int gidx = my + i; // k*ngridxy + j*ngridx + i;
          const numtyp2 tq = grid[gidx];
          const numtyp tq_1 = tq.x; //grid[gidx];
          const numtyp tq_2 = tq.y; //grid[gidx+1];
          t0_1 += tq_1*tha1.x;
          t1_1 += tq_1*tha1.y;
          t2_1 += tq_1*tha1.z;
          t0_2 += tq_2*tha1.x;
          t1_2 += tq_2*tha1.y;
          t2_2 += tq_2*tha1.z;
          t3 += (tq_1+tq_2)*tha1.w;
          i++;
        }

        tu00_1 += t0_1*u0;
        tu10_1 += t1_1*u0;
        tu01_1 += t0_1*u1;
        tu20_1 += t2_1*u0;
        tu11_1 += t1_1*u1;
        tu02_1 += t0_1*u2;
        tu00_2 += t0_2*u0;
        tu10_2 += t1_2*u0;
        tu01_2 += t0_2*u1;
        tu20_2 += t2_2*u0;
        tu11_2 += t1_2*u1;
        tu02_2 += t0_2*u2;
        numtyp t0 = t0_1 + t0_2;
        numtyp t1 = t1_1 + t1_2;
        numtyp t2 = t2_1 + t2_2;
        tu00 += t0*u0;
        tu10 += t1*u0;
        tu01 += t0*u1;
        tu20 += t2*u0;
        tu11 += t1*u1;
        tu02 += t0*u2;
        tu30 += t3*u0;
        tu21 += t2*u1;
        tu12 += t1*u2;
        tu03 += t0*u3;
        j++;
      }

      tuv100_1 += tu10_1*v0;
      tuv010_1 += tu01_1*v0;
      tuv001_1 += tu00_1*v1;
      tuv200_1 += tu20_1*v0;
      tuv020_1 += tu02_1*v0;
      tuv002_1 += tu00_1*v2;
      tuv110_1 += tu11_1*v0;
      tuv101_1 += tu10_1*v1;
      tuv011_1 += tu01_1*v1;
      tuv100_2 += tu10_2*v0;
      tuv010_2 += tu01_2*v0;
      tuv001_2 += tu00_2*v1;
      tuv200_2 += tu20_2*v0;
      tuv020_2 += tu02_2*v0;
      tuv002_2 += tu00_2*v2;
      tuv110_2 += tu11_2*v0;
      tuv101_2 += tu10_2*v1;
      tuv011_2 += tu01_2*v1;
      tuv000 += tu00*v0;
      tuv100 += tu10*v0;
      tuv010 += tu01*v0;
      tuv001 += tu00*v1;
      tuv200 += tu20*v0;
      tuv020 += tu02*v0;
      tuv002 += tu00*v2;
      tuv110 += tu11*v0;
      tuv101 += tu10*v1;
      tuv011 += tu01*v1;
      tuv300 += tu30*v0;
      tuv030 += tu03*v0;
      tuv003 += tu00*v3;
      tuv210 += tu21*v0;
      tuv201 += tu20*v1;
      tuv120 += tu12*v0;
      tuv021 += tu02*v1;
      tuv102 += tu10*v2;
      tuv012 += tu01*v2;
      tuv111 += tu11*v1;
      k++;
    }

    int idx;
    acctyp fdip_buf[20];

    fdip_buf[0] = (numtyp)0.0;
    fdip_buf[1] = tuv100_1;
    fdip_buf[2] = tuv010_1;
    fdip_buf[3] = tuv001_1;
    fdip_buf[4] = tuv200_1;
    fdip_buf[5] = tuv020_1;
    fdip_buf[6] = tuv002_1;
    fdip_buf[7] = tuv110_1;
    fdip_buf[8] = tuv101_1;
    fdip_buf[9] = tuv011_1;
    idx = ii;    
    for (int m = 0; m < 10; m++) {
      fdip_phi1[idx] = fdip_buf[m];
      idx += inum;
    }

    fdip_buf[0] = (numtyp)0.0;
    fdip_buf[1] = tuv100_2;
    fdip_buf[2] = tuv010_2;
    fdip_buf[3] = tuv001_2;
    fdip_buf[4] = tuv200_2;
    fdip_buf[5] = tuv020_2;
    fdip_buf[6] = tuv002_2;
    fdip_buf[7] = tuv110_2;
    fdip_buf[8] = tuv101_2;
    fdip_buf[9] = tuv011_2;
    idx = ii;    
    for (int m = 0; m < 10; m++) {
      fdip_phi2[idx] = fdip_buf[m];
      idx += inum;
    }

    fdip_buf[0] = tuv000;
    fdip_buf[1] = tuv100;
    fdip_buf[2] = tuv010;
    fdip_buf[3] = tuv001;
    fdip_buf[4] = tuv200;
    fdip_buf[5] = tuv020;
    fdip_buf[6] = tuv002;
    fdip_buf[7] = tuv110;
    fdip_buf[8] = tuv101;
    fdip_buf[9] = tuv011;
    fdip_buf[10] = tuv300;
    fdip_buf[11] = tuv030;
    fdip_buf[12] = tuv003;
    fdip_buf[13] = tuv210;
    fdip_buf[14] = tuv201;
    fdip_buf[15] = tuv120;
    fdip_buf[16] = tuv021;
    fdip_buf[17] = tuv102;
    fdip_buf[18] = tuv012;
    fdip_buf[19] = tuv111;
    idx = ii;    
    for (int m = 0; m < 20; m++) {
      fdip_sum_phi[idx] = fdip_buf[m];
      idx += inum;
    }
  }
}

/* ----------------------------------------------------------------------
   fphi_mpole = multipole potential from grid
   fphi_mpole extracts the permanent multipole potential from
   the particle mesh Ewald grid
------------------------------------------------------------------------- */

__kernel void k_amoeba_fphi_mpole(const __global numtyp4 *restrict thetai1,
                          const __global numtyp4 *restrict thetai2,
                          const __global numtyp4 *restrict thetai3,
                          const __global int *restrict igrid,
                          const __global numtyp2 *restrict grid,
                          __global acctyp *restrict fphi,
                          const int bsorder, const int inum, const numtyp felec,
                          const int nzlo_out, const int nylo_out,
                          const int nxlo_out, const int ngridxy,
                          const int ngridx)
{
  int tid=THREAD_ID_X;
  int ii=tid+BLOCK_ID_X*BLOCK_SIZE_X;

  if (ii<inum) {

    int nlpts = (bsorder-1) / 2;
    
    int istart = fast_mul(ii,4);
    int igridx = igrid[istart];
    int igridy = igrid[istart+1];
    int igridz = igrid[istart+2];
    
    // now istart is used to index thetai1, thetai2 and thetai3
    istart = fast_mul(ii,bsorder);

    // extract the permanent multipole field at each site

    numtyp tuv000 = (numtyp)0.0;
    numtyp tuv001 = (numtyp)0.0;
    numtyp tuv010 = (numtyp)0.0;
    numtyp tuv100 = (numtyp)0.0;
    numtyp tuv200 = (numtyp)0.0;
    numtyp tuv020 = (numtyp)0.0;
    numtyp tuv002 = (numtyp)0.0;
    numtyp tuv110 = (numtyp)0.0;
    numtyp tuv101 = (numtyp)0.0;
    numtyp tuv011 = (numtyp)0.0;
    numtyp tuv300 = (numtyp)0.0;
    numtyp tuv030 = (numtyp)0.0;
    numtyp tuv003 = (numtyp)0.0;
    numtyp tuv210 = (numtyp)0.0;
    numtyp tuv201 = (numtyp)0.0;
    numtyp tuv120 = (numtyp)0.0;
    numtyp tuv021 = (numtyp)0.0;
    numtyp tuv102 = (numtyp)0.0;
    numtyp tuv012 = (numtyp)0.0;
    numtyp tuv111 = (numtyp)0.0;

    int k = (igridz - nzlo_out) - nlpts;
    for (int kb = 0; kb < bsorder; kb++) {
      int i3 = istart + kb;
      numtyp4 tha3 = thetai3[i3];
      numtyp v0 = tha3.x;
      numtyp v1 = tha3.y;
      numtyp v2 = tha3.z;
      numtyp v3 = tha3.w;
      numtyp tu00 = (numtyp)0.0;
      numtyp tu10 = (numtyp)0.0;
      numtyp tu01 = (numtyp)0.0;
      numtyp tu20 = (numtyp)0.0;
      numtyp tu11 = (numtyp)0.0;
      numtyp tu02 = (numtyp)0.0;
      numtyp tu30 = (numtyp)0.0;
      numtyp tu21 = (numtyp)0.0;
      numtyp tu12 = (numtyp)0.0;
      numtyp tu03 = (numtyp)0.0;

      int j = (igridy - nylo_out) - nlpts;
      for (int jb = 0; jb < bsorder; jb++) {
        int i2 = istart + jb;
        numtyp4 tha2 = thetai2[i2];
        numtyp u0 = tha2.x;
        numtyp u1 = tha2.y;
        numtyp u2 = tha2.z;
        numtyp u3 = tha2.w;
        numtyp t0 = (numtyp)0.0;
        numtyp t1 = (numtyp)0.0;
        numtyp t2 = (numtyp)0.0;
        numtyp t3 = (numtyp)0.0;

        int i = (igridx - nxlo_out) - nlpts;
        for (int ib = 0; ib < bsorder; ib++) {
          int i1 = istart + ib;
          numtyp4 tha1 = thetai1[i1];
          int gidx = k*ngridxy + j*ngridx + i;
          numtyp tq = grid[gidx].x;
          t0 += tq*tha1.x;
          t1 += tq*tha1.y;
          t2 += tq*tha1.z;
          t3 += tq*tha1.w;
          i++;
        }

        tu00 += t0*u0;
        tu10 += t1*u0;
        tu01 += t0*u1;
        tu20 += t2*u0;
        tu11 += t1*u1;
        tu02 += t0*u2;
        tu30 += t3*u0;
        tu21 += t2*u1;
        tu12 += t1*u2;
        tu03 += t0*u3;
        j++;
      }

      tuv000 += tu00*v0;
      tuv100 += tu10*v0;
      tuv010 += tu01*v0;
      tuv001 += tu00*v1;
      tuv200 += tu20*v0;
      tuv020 += tu02*v0;
      tuv002 += tu00*v2;
      tuv110 += tu11*v0;
      tuv101 += tu10*v1;
      tuv011 += tu01*v1;
      tuv300 += tu30*v0;
      tuv030 += tu03*v0;
      tuv003 += tu00*v3;
      tuv210 += tu21*v0;
      tuv201 += tu20*v1;
      tuv120 += tu12*v0;
      tuv021 += tu02*v1;
      tuv102 += tu10*v2;
      tuv012 += tu01*v2;
      tuv111 += tu11*v1;
      k++;
    }

    numtyp buf[20];
    buf[0] = tuv000;
    buf[1] = tuv100;
    buf[2] = tuv010;
    buf[3] = tuv001;
    buf[4] = tuv200;
    buf[5] = tuv020;
    buf[6] = tuv002;
    buf[7] = tuv110;
    buf[8] = tuv101;
    buf[9] = tuv011;
    buf[10] = tuv300;
    buf[11] = tuv030;
    buf[12] = tuv003;
    buf[13] = tuv210;
    buf[14] = tuv201;
    buf[15] = tuv120;
    buf[16] = tuv021;
    buf[17] = tuv102;
    buf[18] = tuv012;
    buf[19] = tuv111;

    int idx = ii;    
    for (int m = 0; m < 20; m++) {
      fphi[idx] = felec * buf[m];
      idx += inum;
    }
  }
}

/* ----------------------------------------------------------------------
   scan standard neighbor list and make it compatible with 1-5 neighbors
   if IJ entry is a 1-2,1-3,1-4 neighbor then adjust offset to SBBITS15
   else scan special15 to see if a 1-5 neighbor and adjust offset to SBBITS15
   else do nothing to IJ entry
------------------------------------------------------------------------- */

__kernel void k_amoeba_special15(__global int * dev_nbor,
                          const __global int * dev_packed,
                          const __global tagint *restrict tag,
                          const __global int *restrict nspecial15,
                          const __global tagint *restrict special15,
                          const int inum, const int nall, const int nbor_pitch,
                          const int t_per_atom) {
  int tid, ii, offset, n_stride, i;
  atom_info(t_per_atom,ii,tid,offset);

  if (ii<inum) {

    int numj, nbor, nbor_end;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    int n15 = nspecial15[ii];

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int sj=dev_packed[nbor];
      int which = sj >> SBBITS & 3;
      int j = sj & NEIGHMASK;
      tagint jtag = tag[j];

      if (!which) {
        int offset=ii;
        for (int k=0; k<n15; k++) {
          if (special15[offset] == jtag) {
            which = 4;
            break;
          }
          offset += nall;
        }
      }

      if (which) dev_nbor[nbor] = j ^ (which << SBBITS15);
    } // for nbor

  } // if ii
}

__kernel void k_amoeba_short_nbor(const __global numtyp4 *restrict x_,
                                  const __global int * dev_nbor,
                                  const __global int * dev_packed,
                                  __global int * dev_short_nbor,
                                  const numtyp off2,
                                  const int inum, const int nbor_pitch,
                                  const int t_per_atom) {
  __local int n_stride;
  int tid, ii, offset;
  atom_info(t_per_atom,ii,tid,offset);

  if (ii<inum) {
    int nbor, nbor_end;
    int i, numj;
    nbor_info(dev_nbor,dev_packed,nbor_pitch,t_per_atom,ii,offset,i,numj,
              n_stride,nbor_end,nbor);

    numtyp4 ix; fetch4(ix,i,pos_tex); //x_[i];

    int ncount = 0;
    int m = nbor;
    dev_short_nbor[m] = 0;
    int nbor_short = nbor+n_stride;

    for ( ; nbor<nbor_end; nbor+=n_stride) {

      int j=dev_packed[nbor];
      int nj = j;
      j &= NEIGHMASK15;

      numtyp4 jx; fetch4(jx,j,pos_tex); //x_[j];

      // Compute r12
      numtyp delx = ix.x-jx.x;
      numtyp dely = ix.y-jx.y;
      numtyp delz = ix.z-jx.z;
      numtyp rsq = delx*delx+dely*dely+delz*delz;

      if (rsq<off2) {
        dev_short_nbor[nbor_short] = nj;
        nbor_short += n_stride;
        ncount++;
      }
    } // for nbor

    // store the number of neighbors for each thread
    dev_short_nbor[m] = ncount;

  } // if ii
}
